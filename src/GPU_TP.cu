#include "hip/hip_runtime.h"

#include "GPU_TP.hpp"

namespace {

__global__ void convolution(char* InImg, char* OutImg, int ImgWidth, int ImgHeight, char* Mask, int MaskWidth, int MaskHeight)
{
	int idX =
		threadIdx.x
		+ blockIdx.x * blockDim.x;

	int idY =
		threadIdx.y
		+ blockIdx.y * blockDim.y;

	int idGlobal = idY * ImgWidth + idX;

	if (idX < ImgWidth && idY < ImgHeight)
	{
		for (int i = 0; i < MaskHeight; i++)
		{
			for (int j = 0; j < MaskWidth; j++)
			{
				int idMask = i * MaskWidth + j;
				int shift = MaskHeight / 2;
				int idImg = (idY + i - shift) * ImgWidth + (idX + j - shift);
				if ((idY + i - shift) >= 0 && (idY + i - shift) < ImgHeight && (idX + j - shift) >= 0 && (idX + j - shift) < ImgWidth)
				{
					OutImg[idGlobal] += InImg[idImg] * Mask[idMask];
				}
			}
		}
	}
}

void convolution(std::vector<char>& image, const int width, const std::vector<char>& mask, const int widthMask)
{
	int height = image.size() / width;
	int heightMask = mask.size() / widthMask;
	hipError_t err;

	// Allocation de la m�moire sur le GPU
	char* inImage;
	err = hipMalloc(&inImage, width * height * sizeof(char));
	if (err != hipSuccess)
	{
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		return;
	}

	char* outImage;
	err = hipMalloc(&outImage, width * height * sizeof(char));
	if (err != hipSuccess)
	{
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		return;
	}

	// Initialisation de la m�moire sur le GPU � 0
	err = hipMemset(outImage, (char) 0, width * height * sizeof(char));
	if (err != hipSuccess)
	{
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		return;
	}

	char* inMask;
	err = hipMalloc(&inMask, widthMask * heightMask * sizeof(char));
	if (err != hipSuccess)
	{
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		return;
	}

	// Copie des donn�es sur le GPU
	err = hipMemcpy(inImage, image.data(), width * height * sizeof(char), hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		return;
	}

	err = hipMemcpy(inMask, mask.data(), widthMask * heightMask * sizeof(char), hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		return;
	}

	
	// D�finition de la taille des blocs et de la grille
	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);
	// Appel du kernel
	convolution << <numBlocks, threadsPerBlock >> > (inImage, outImage, width, height, inMask, widthMask, heightMask);
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		return;
	}

	// Copie des donn�es du GPU vers le CPU
	err = hipMemcpy(image.data(), outImage, width * height * sizeof(char), hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		return;
	}

	// Lib�ration de la m�moire
	hipFree(inImage);
	hipFree(outImage);
	hipFree(inMask);
}

__global__ void convolution(char** InImg, char** OutImg, int ImgWidth, int ImgHeight, char* Mask, int MarkWidth, int MaskHeight)
{
	// TODO
}

std::vector<int> convolution(std::vector<int>& image, const int width, const std::vector<int>& mask, const int widthMask)
{
	return std::vector<int>();
}

} // namespace

void runOnGPU()
{
	int imageWidth, imageHeight, maskWidth, maskHeight;
	std::cout << "Enter the image width: ";
	std::cin >> imageWidth;
	std::cout << "Enter the image height: ";
	std::cin >> imageHeight;
	std::cout << "Enter the mask width: ";
	std::cin >> maskWidth;
	std::cout << "Enter the mask height: ";
	std::cin >> maskHeight;

	std::vector<char> image;
	// Initialisation de l'image � i % 255
	for (int i = 0; i < imageWidth * imageHeight; i++)
	{
		image.push_back(i % 255);
	}

	// Affichage de l'image le premier carr� de 5x5 pixels de l'image
	std::cout << "Image before convolution: " << std::endl;
	for (int i = 0; i < 5; i++)
	{
		for (int j = 0; j < 5; j++)
		{
			std::cout << (int)image[i * imageWidth + j] << " ";
		}
		std::cout << std::endl;
	}

	// Masque
	std::vector<char> mask = {
		1, 0, 0,
		0, 0, 0,
		0, 0, 0
	};

	// Convolution
	convolution(image, imageWidth, mask, maskWidth);

	// Affichage de l'image le premier carr� de 5x5 pixels de l'image
	std::cout << "Image after convolution: " << std::endl;
	for (int i = 0; i < 5; i++)
	{
		for (int j = 0; j < 5; j++)
		{
			std::cout << (int)image[i * imageWidth + j] << " ";
		}
		std::cout << std::endl;
	}
}
