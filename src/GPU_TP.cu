#include "hip/hip_runtime.h"

#include "GPU_TP.hpp"

namespace {

	__global__ void convolution(unsigned char* InImg, unsigned char* OutImg, int ImgWidth, int ImgHeight, char* Mask, int MaskWidth, int MaskHeight)
	{
		int idX =
			threadIdx.x
			+ blockIdx.x * blockDim.x;

		int idY =
			threadIdx.y
			+ blockIdx.y * blockDim.y;

		int idGlobal = idY * ImgWidth + idX;

		// Copie du masque dans la m�moire partag�e MaskWidth * MaskHeight
		extern __shared__ char sharedMask[];
		if (threadIdx.x < MaskWidth && threadIdx.y < MaskHeight)
		{
			sharedMask[threadIdx.y * MaskWidth + threadIdx.x] = Mask[threadIdx.y * MaskWidth + threadIdx.x];
		}

		__syncthreads();

		if (idX < ImgWidth && idY < ImgHeight)
		{
			for (int i = 0; i < MaskHeight; i++)
			{
				for (int j = 0; j < MaskWidth; j++)
				{
					int idMask = i * MaskWidth + j;
					int shift = MaskHeight / 2;
					int idImg = (idY + i - shift) * ImgWidth + (idX + j - shift);
					if ((idY + i - shift) >= 0 && (idY + i - shift) < ImgHeight && (idX + j - shift) >= 0 && (idX + j - shift) < ImgWidth)
					{
						OutImg[idGlobal] += InImg[idImg] * sharedMask[idMask];
					}
				}
			}
		}
	}

	__global__ void convolution(int* InImg, int* OutImg, int ImgWidth, int ImgHeight, int* Mask, int MaskWidth, int MaskHeight)
	{
		int idX =
			threadIdx.x
			+ blockIdx.x * blockDim.x;

		int idY =
			threadIdx.y
			+ blockIdx.y * blockDim.y;

		int idGlobal = idY * ImgWidth + idX;

		// Copie du masque dans la m�moire partag�e MaskWidth * MaskHeight
		extern __shared__ char sharedMask[];
		if (threadIdx.x < MaskWidth && threadIdx.y < MaskHeight)
		{
			sharedMask[threadIdx.y * MaskWidth + threadIdx.x] = Mask[threadIdx.y * MaskWidth + threadIdx.x];
		}

		__syncthreads();

		// On v�rifie que le pixel est dans l'image
		if (idX < ImgWidth && idY < ImgHeight)
		{
			// Initialisation de la valeur du pixel
			unsigned char R = 0, G = 0, B = 0;
			// Parcours du masque
			for (int i = 0; i < MaskHeight; i++)
			{
				for (int j = 0; j < MaskWidth; j++)
				{
					int idMask = i * MaskWidth + j;
					int shift = MaskHeight / 2;
					int idImg = (idY + i - shift) * ImgWidth + (idX + j - shift);
					if ((idY + i - shift) >= 0 && (idY + i - shift) < ImgHeight && (idX + j - shift) >= 0 && (idX + j - shift) < ImgWidth)
					{
						R += (InImg[idImg] >> 24) & 0xFF * sharedMask[idMask];
						G += (InImg[idImg] >> 16) & 0xFF * sharedMask[idMask];
						B += (InImg[idImg] >> 8) & 0xFF * sharedMask[idMask];
					}
				}
			}
			// Ajout de la valeur du pixel dans le r�sultat
			OutImg[idGlobal] = (R << 24) | (G << 16) | (B << 8);
		}
	}

	std::vector<unsigned char> convolution(std::vector<unsigned char>& image, const int width, const std::vector<char>& mask, const int widthMask)
	{
		std::vector<unsigned char> result(image.size(), 0);

		// V�rifie que la taille de l'image est un multiple de la largeur de l'image
		if (image.size() % width != 0)
		{
			std::cerr << "Error: image width is not a multiple of the image size" << std::endl;
			return result;
		}
		// V�rifie que la taille du masque est un multiple de la largeur du masque
		if (mask.size() % widthMask != 0)
		{
			std::cerr << "Error: mask width is not a multiple of the mask size" << std::endl;
			return result;
		}

		int height = image.size() / width;
		int heightMask = mask.size() / widthMask;
		hipError_t err;

		// Allocation de la m�moire sur le GPU
		unsigned char* inImage;
		err = hipMalloc(&inImage, width * height * sizeof(unsigned char));
		if (err != hipSuccess)
		{
			std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
			return result;
		}

		unsigned char* outImage;
		err = hipMalloc(&outImage, width * height * sizeof(unsigned char));
		if (err != hipSuccess)
		{
			std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
			return result;
		}

		// Initialisation de la m�moire sur le GPU � 0
		err = hipMemset(outImage, (unsigned char)0, width * height * sizeof(unsigned char));
		if (err != hipSuccess)
		{
			std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
			return result;
		}

		char* inMask;
		err = hipMalloc(&inMask, widthMask * heightMask * sizeof(char));
		if (err != hipSuccess)
		{
			std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
			return result;
		}

		// Copie des donn�es sur le GPU
		err = hipMemcpy(inImage, image.data(), width * height * sizeof(unsigned char), hipMemcpyHostToDevice);
		if (err != hipSuccess)
		{
			std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
			return result;
		}

		err = hipMemcpy(inMask, mask.data(), widthMask * heightMask * sizeof(unsigned char), hipMemcpyHostToDevice);
		if (err != hipSuccess)
		{
			std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
			return result;
		}


		// D�finition de la taille des blocs et de la grille
		dim3 threadsPerBlock(16, 16);
		dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);
		// Appel du kernel convolution avec la taille du masque en m�moire partag�e
		convolution << <numBlocks, threadsPerBlock, widthMask* heightMask * sizeof(char) >> > (inImage, outImage, width, height, inMask, widthMask, heightMask);
		err = hipGetLastError();
		if (err != hipSuccess)
		{
			std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
			return result;
		}

		// Copie des donn�es du GPU vers le CPU
		err = hipMemcpy(result.data(), outImage, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);
		if (err != hipSuccess)
		{
			std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
			return result;
		}

		// Lib�ration de la m�moire
		hipFree(inImage);
		hipFree(outImage);
		hipFree(inMask);

		return result;
	}

	std::vector<int> convolution(std::vector<int>& image, const int width, const std::vector<int>& mask, const int widthMask)
	{
		std::vector<int> result(image.size(), 0);

		// V�rifie que la taille de l'image est un multiple de la largeur de l'image
		if (image.size() % width != 0)
		{
			std::cerr << "Error: image width is not a multiple of the image size" << std::endl;
			return result;
		}
		// V�rifie que la taille du masque est un multiple de la largeur du masque
		if (mask.size() % widthMask != 0)
		{
			std::cerr << "Error: mask width is not a multiple of the mask size" << std::endl;
			return result;
		}

		int height = image.size() / width;
		int heightMask = mask.size() / widthMask;
		hipError_t err;

		// Allocation de la m�moire sur le GPU
		int* inImage;
		err = hipMalloc(&inImage, width * height * sizeof(int));
		if (err != hipSuccess)
		{
			std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
			return result;
		}

		int* outImage;
		err = hipMalloc(&outImage, width * height * sizeof(int));
		if (err != hipSuccess)
		{
			std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
			return result;
		}

		// Initialisation de la m�moire sur le GPU � 0
		err = hipMemset(outImage, (int)0, width * height * sizeof(int));
		if (err != hipSuccess)
		{
			std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
			return result;
		}

		int* inMask;
		err = hipMalloc(&inMask, widthMask * heightMask * sizeof(char));
		if (err != hipSuccess)
		{
			std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
			return result;
		}

		// Copie des donn�es sur le GPU
		err = hipMemcpy(inImage, image.data(), width * height * sizeof(int), hipMemcpyHostToDevice);
		if (err != hipSuccess)
		{
			std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
			return result;
		}

		err = hipMemcpy(inMask, mask.data(), widthMask * heightMask * sizeof(int), hipMemcpyHostToDevice);
		if (err != hipSuccess)
		{
			std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
			return result;
		}


		// D�finition de la taille des blocs et de la grille
		dim3 threadsPerBlock(16, 16);
		dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);
		// Appel du kernel convolution avec la taille du masque en m�moire partag�e
		convolution << <numBlocks, threadsPerBlock, widthMask* heightMask * sizeof(int) >> > (inImage, outImage, width, height, inMask, widthMask, heightMask);
		err = hipGetLastError();
		if (err != hipSuccess)
		{
			std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
			return result;
		}

		// Copie des donn�es du GPU vers le CPU
		err = hipMemcpy(result.data(), outImage, width * height * sizeof(unsigned int), hipMemcpyDeviceToHost);
		if (err != hipSuccess)
		{
			std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
			return result;
		}

		// Lib�ration de la m�moire
		hipFree(inImage);
		hipFree(outImage);
		hipFree(inMask);

		return result;
	}

} // namespace

void runOnGPU_GREY()
{
	int imageWidth, imageHeight, maskWidth;
	std::cout << "Enter the image width: ";
	std::cin >> imageWidth;
	std::cout << "Enter the image height: ";
	std::cin >> imageHeight;

	std::vector<unsigned char> image;
	// Initialisation de l'image � i % 255 pour les 3 canaux RGB
	for (int i = 0; i < imageWidth * imageHeight; i++)
	{
		image.push_back(i);
	}

	// Affichage de l'image le premier carr� de 5x5 pixels de l'image
	std::cout << "Image before convolution: " << std::endl;
	for (int i = 0; i < 10; i++)
	{
		for (int j = 0; j < 10; j++)
		{
			std::cout << std::setw(3) << (int)image[i * imageWidth + j] << " ";
		}
		std::cout << std::endl;
	}

	// Masque
	maskWidth = 3;
	std::vector<char> mask = {
		1, 0, 0,
		0, 0, 0,
		0, 0, 0
	};

	// Convolution
	std::vector<unsigned char> result = convolution(image, imageWidth, mask, maskWidth);
	std::cout << "Result size: " << result.size() << std::endl;

	// Affichage de l'image le premier carr� de 5x5 pixels de l'image
	std::cout << "Image after convolution: " << std::endl;
	for (int i = 0; i < 10; i++)
	{
		for (int j = 0; j < 10; j++)
		{
			std::cout << std::setw(3) << (int)result[i * imageWidth + j] << " ";
		}
		std::cout << std::endl;
	}
}

void runOnGPU_RGB()
{
	int imageWidth, imageHeight, maskWidth;
	std::cout << "Enter the image width: ";
	std::cin >> imageWidth;
	std::cout << "Enter the image height: ";
	std::cin >> imageHeight;

	std::vector<int> image;
	// Initialisation de l'image � i % 255 pour les 3 canaux RGB
	for (int i = 0; i < imageWidth * imageHeight; i++)
	{
		image.push_back((i << 24) | (i << 16) | (i << 8));
	}

	// Affichage de l'image le premier carr� de 5x5 pixels canal par canal
	std::cout << "Image after convolution: " << std::endl;
	for (int i = 0; i < 5; i++)
	{
		for (int j = 0; j < 5; j++)
		{
			std::cout << std::setw(3) << (int)((image[i * imageWidth + j] >> 24) & 0xFF) << "|";
			std::cout << std::setw(3) << (int)((image[i * imageWidth + j] >> 16) & 0xFF) << "|";
			std::cout << std::setw(3) << (int)((image[i * imageWidth + j] >> 8) & 0xFF) << "\t";
		}
		std::cout << std::endl;
	}

	// Masque
	maskWidth = 3;
	std::vector<int> mask = {
		1, 0, 0,
		0, 0, 0,
		0, 0, 0
	};

	// Convolution
	std::vector<int> result = convolution(image, imageWidth, mask, maskWidth);
	std::cout << "Result size: " << result.size() << std::endl;

	// Affichage de l'image le premier carr� de 5x5 pixels canal par canal
	std::cout << "Image after convolution: " << std::endl;
	for (int i = 0; i < 5; i++)
	{
		for (int j = 0; j < 5; j++)
		{
			std::cout << std::setw(3) << (int)((result[i * imageWidth + j] >> 24) & 0xFF) << "|";
			std::cout << std::setw(3) << (int)((result[i * imageWidth + j] >> 16) & 0xFF) << "|";
			std::cout << std::setw(3) << (int)((result[i * imageWidth + j] >> 8) & 0xFF) << "\t";
		}
		std::cout << std::endl;
	}
}
